#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""
#include "deviceFunctions.h"

__global__ void gatherScan(int* queueIn, int*queueOut, int* C, int* R,
	int* neighboursPrefixSum,
	int* visited, int* totalNeighbours,
	int* neighbourCountsOut, int newColor, int startingColor)
{
	__shared__ int neighbours[THREAD_NUM];

	if (*totalNeighbours == 1 && threadIdx.x > 0)
	{
		//neighbours[0] = 1;
	}

	int v = queueIn[threadIdx.x];
	int r, rEnd, index;
	if (v != -1)
	{
		r = R[v];// rArrIn[v];					// index of first v's neighbour from C array
		rEnd = R[v + 1];// rEndArrayIn[v];			// index of last v's neighbour from C array
		index = neighboursPrefixSum[threadIdx.x];
		visited[v] = newColor;
	}
	if (v == 13)
	{
		//neighbours[0] = 1;
	}
	int blockProgress = 0;
	int remain;
	while ((remain = *totalNeighbours - blockProgress) > 0)
	{
		if (v != -1 && index < *totalNeighbours && index >= 0)
		{
			// put vertex v's neighbours to shared memory
			while ((index < blockProgress + THREAD_NUM)
				&& (r < rEnd))
			{
				neighbours[index - blockProgress] = r; // r shows where current v's neighbour is in C array
				index++;
				r++;
			}
		}
		__syncthreads();
		// each thread gets a vertex from shared memory
		if (threadIdx.x < remain && threadIdx.x < THREAD_NUM) {
			int v = C[neighbours[threadIdx.x]];
			if (visited[v] != startingColor)
			{
				v = -1;
				neighbourCountsOut[blockProgress + threadIdx.x] = 0;
			}
			else
			{
				int newR = R[v];
				int newREnd = R[v + 1];
				neighbourCountsOut[blockProgress + threadIdx.x] = newREnd - newR;
			}
			queueOut[blockProgress + threadIdx.x] = v;
		}
		blockProgress += THREAD_NUM;
		__syncthreads();
	}
}