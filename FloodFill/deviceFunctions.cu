#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""
#include "deviceFunctions.h"

__global__ void gatherScan(int* queueIn, int*queueOut, int* C, int* R,
	int* neighboursPrefixSum, int* visited, int* totalNeighbours,
	int* neighbourCountsOut, int newColor, int startingColor)
{
	__shared__ int neighbours[THREAD_NUM];

	int v = queueIn[threadIdx.x];
	int r, rEnd, index;
	if (v != -1)
	{
		r = R[v];					// index of first v's neighbour from C array
		rEnd = R[v + 1];			// index of last v's neighbour from C array
		index = neighboursPrefixSum[threadIdx.x];	// index at which this thread will start putting v's neighbours in queueOut
		visited[v] = newColor;		
	}

	int blockProgress = 0;	// number of vertices put in queueOut in all previous iteration of main while loop
	int remain;
	while ((remain = *totalNeighbours - blockProgress) > 0)
	{
		if (v != -1 && index < *totalNeighbours && index >= 0)	// if index is out of range <0, totalNeighbours>, this thread should be idle
		{
			// put vertex v's neighbours to shared memory
			while ((index < blockProgress + THREAD_NUM)
				&& (r < rEnd))
			{
				neighbours[index - blockProgress] = r; // r shows where currently viewed v's neighbour is in C array
				index++;
				r++;
			}
		}
		__syncthreads();
		// each thread gets a vertex from shared memory
		if (threadIdx.x < remain && threadIdx.x < THREAD_NUM) {
			int n = C[neighbours[threadIdx.x]]; // v's neighbour
			if (visited[n] != startingColor)
			{
				n = -1;				// we don't want to process n in next bfs iteration
				neighbourCountsOut[blockProgress + threadIdx.x] = 0;
			}
			else
			{
				int newR = R[n];
				int newREnd = R[n + 1];
				neighbourCountsOut[blockProgress + threadIdx.x] = newREnd - newR; // save number of n's neighbours
			}
			queueOut[blockProgress + threadIdx.x] = n;
		}
		blockProgress += THREAD_NUM;
		__syncthreads();
	}
}