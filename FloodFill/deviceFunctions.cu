#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include "deviceFunctions.h"

__global__ void GatherScan(int* queueIn, int*queueOut, int* C, int* rArr,
	int* rEndArray, int* neighboursPrefixSum,
	bool* visited, int totalNeighbours)
{
	__shared__ int neighbours[THREAD_NUM];

	int v = queueIn[threadIdx.x];
	int r = rArr[v];					// index of first v's neighbour from C array
	int rEnd = rEndArray[v];			// index of last v's neighbour from C array
	int index = neighboursPrefixSum[v];

	visited[v] = true;
	int blockProgress = 0;
	int remain;
	while ((remain = totalNeighbours - blockProgress) > 0)
	{
		// put vertex v's neighbours to shared memory
		while ((index < blockProgress + THREAD_NUM)
			&& (r < rEnd))
		{
			neighbours[index - blockProgress] = r; // r shows where current v's neighbour is in C array
			index++;
			r++;
		}
		__syncthreads();
		// each thread gets a vertex from shared memory
		if (threadIdx.x < remain && threadIdx.x < THREAD_NUM) {
			queueOut[blockProgress + threadIdx.x] = C[neighbours[threadIdx.x]];
		}
		blockProgress += THREAD_NUM;
		__syncthreads();
	}
}