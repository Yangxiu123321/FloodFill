
#include "hip/hip_runtime.h"
#include ""
#include <thrust/scan.h>

#include <stdio.h>
#include <stdlib.h>

#include "hostFunctions.h"

void bfs(int** array, int height, int width, int xStarting, int yStarting, int newColor)
{
	int verticesCount = height * width;
	int oldColor = array[yStarting][xStarting];

	int* rowRangeOffsets = (int*)malloc((verticesCount + 1) * sizeof(int));
	int* neighboursCounts = (int*)malloc(verticesCount * sizeof(int));
	int v = 0;

	for (int y = 0; y < height; y++)
	{
		for (int x = 0; x < width; x++, v++)
		{
			int nei = 0; // neighbours count
			if (array[y][x] == oldColor)
			{
				if (y > 0 && array[y - 1][x] == oldColor) nei++;
				if (y < height - 1 && array[y + 1][x] == oldColor) nei++;
				if (x > 0 && array[y][x - 1] == oldColor) nei++;
				if (x < width - 1 && array[y][x + 1] == oldColor) nei++;
			}
			neighboursCounts[v] = nei;
		}
	}
	rowRangeOffsets[0] = 0;
	thrust::inclusive_scan(neighboursCounts, neighboursCounts + verticesCount, rowRangeOffsets + 1);
	/*for (int i = 0; i <= verticesCount; i++)
		printf("%d ", rowRangeOffsets[i]);
	printf("\n");*/
	int* columnIndices = (int*)malloc(rowRangeOffsets[verticesCount] * sizeof(int));

	// construct array C with neighbours of every vertex
	for (int y = 0, index = 0; y < height; y++)
	{
		for (int x = 0; x < width; x++, v++)
		{
			if (array[y][x] == oldColor)
			{
				if (y > 0 && array[y - 1][x] == oldColor) columnIndices[index++] = v;
				if (y < height - 1 && array[y + 1][x] == oldColor) columnIndices[index++] = v;
				if (x > 0 && array[y][x - 1] == oldColor) columnIndices[index++] = v;;
				if (x < width - 1 && array[y][x + 1] == oldColor) columnIndices[index++] = v;
			}
		}
	}

}

int getEdgesCount(int rows, int cols)
{
	return 4 * (rows - 2) * (cols - 2) + 3 * 2 * (rows - 2) + 3 * 2 * (cols - 2) + 8;
}

void prepareArrays(int* inputArr, int** R, int** C, int rows, int cols)
{
	int verticesCount = rows * cols;
	*R = (int*)malloc((verticesCount + 1) * sizeof(int));
	int edgesCount = getEdgesCount(rows, cols);

	*C = (int*)malloc(edgesCount * sizeof(int));

	for (int i = 0, v = 0; i < edgesCount; v++)
	{
		int x = v % cols;
		int y = v / cols;

		if (x == 0 && y == 0)
		{
			(*C)[i++] = v + 1;
			(*C)[i++] = v + cols;
			(*R)[v] = 2;
		}
		else if (x == 0 && y == rows - 1)
		{
			(*C)[i++] = v + 1;
			(*C)[i++] = v - cols;
			(*R)[v] = 2;
		}
		else if (x == cols - 1 && y == 0)
		{
			(*C)[i++] = v - 1;
			(*C)[i++] = v + cols;
			(*R)[v] = 2;
		}
		else if (x == cols - 1 && y == rows - 1)
		{
			(*C)[i++] = v - 1;
			(*C)[i++] = v - cols;
			(*R)[v] = 2;
		}
		else if (x == 0)
		{
			(*C)[i++] = v + 1;
			(*C)[i++] = v + cols;
			(*C)[i++] = v - cols;
			(*R)[v] = 3;
		}
		else if (x == cols - 1)
		{
			(*C)[i++] = v - 1;
			(*C)[i++] = v + cols;
			(*C)[i++] = v - cols;
			(*R)[v] = 3;
		}
		else if (y == 0)
		{
			(*C)[i++] = v - 1;
			(*C)[i++] = v + 1;
			(*C)[i++] = v + cols;
			(*R)[v] = 3;
		}
		else if (y == rows - 1)
		{
			(*C)[i++] = v - 1;
			(*C)[i++] = v + 1;
			(*C)[i++] = v - cols;
			(*R)[v] = 3;
		}
		else
		{
			(*C)[i++] = v - 1;
			(*C)[i++] = v + 1;
			(*C)[i++] = v + cols;
			(*C)[i++] = v - cols;
			(*R)[v] = 4;
		}

	}
	(*R)[verticesCount] = 0;
	thrust::exclusive_scan((*R), (*R) + verticesCount + 1, (*R));

	//for (int i = 0; i < verticesCount + 1; i++)
	//	printf("%d ", (*R)[i]);
	//printf("\n");

	//for (int i = 0; i < edgesCount; i++)
	//	printf("%d ", (*C)[i]);
	//printf("\n");
}

hipError_t deviceMalloc(int** dest, int length)
{
	hipError_t cudaStatus = hipMalloc((void**)dest, length * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
	}
	return cudaStatus;
}

hipError_t deviceMemcpy(int* dest, int* source, int length, hipMemcpyKind direction)
{
	hipError_t cudaStatus = hipMemcpy(dest, source, length * sizeof(int), direction);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
	}
	return cudaStatus;
}

//int main()
//{
//
//	hipError_t cudaStatus;// = addWithCuda(c, a, b, arraySize);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addWithCuda failed!");
//		return 1;
//	}
//	
//
//	// hipDeviceReset must be called before exiting in order for profiling and
//	// tracing tools such as Nsight and Visual Profiler to show complete traces.
//	cudaStatus = hipDeviceReset();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceReset failed!");
//		return 1;
//	}
//
//	return 0;
//}
