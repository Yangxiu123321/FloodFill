#include "floodFillWithCpu.h"
#include "hostFunctions.h"
#include "deviceFunctions.h"
#include "floodFillWithGpu.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/scan.h>


int* FloodFillWithGPU(int* arr, int rows, int cols, int xStarting, int yStarting, int newColor)
{
	int* r;
	int* c;
	int startingVertex = yStarting * cols + xStarting;
	int startingColor = arr[startingVertex];
	int edgesCount = getEdgesCount(rows, cols);
	int verticesCount = rows * cols;

	prepareArrays(arr, &r, &c, rows, cols);

	int* queueIn = (int*)malloc(THREAD_NUM * sizeof(int));
	int* queueOut = (int*)malloc(THREAD_NUM * sizeof(int));
	int* neighboursPrefixSum = (int*)malloc(THREAD_NUM * sizeof(int));
	int* visited = (int*)malloc(rows * cols * sizeof(int));
	int totalNeighbours;
	int* neighbourCounts = (int*)malloc(THREAD_NUM * sizeof(int));

	for (int i = 0; i < THREAD_NUM; i++)
	{
		queueIn[i] = -1;
		queueOut[i] = 0;
		neighboursPrefixSum[i] = 0;
	}
	queueIn[0] = startingVertex;
	neighboursPrefixSum[0] = r[startingVertex + 1] - r[startingVertex];

	for (int i = 0; i < rows * cols; i++)
		visited[i] = arr[i];


	thrust::exclusive_scan(neighboursPrefixSum, neighboursPrefixSum + THREAD_NUM, neighboursPrefixSum);
	totalNeighbours = neighboursPrefixSum[THREAD_NUM - 1];

	hipError_t cudaStatus;

#pragma region Device arrays

	int* dev_c = 0;
	int* dev_r;
	int* dev_queueIn;
	int* dev_queueOut;
	int* dev_neighboursPrefixSum;
	int* dev_visited = 0;
	int* dev_totalNeighbours;
	int* dev_neighbourCounts;

#pragma endregion

#pragma region Mallocs
	cudaStatus = deviceMalloc(&dev_c, edgesCount);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_c, c, edgesCount, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_r, (verticesCount + 1));
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_r, r, (verticesCount + 1), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_queueIn, THREAD_NUM);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_queueIn, queueIn, THREAD_NUM, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_queueOut, THREAD_NUM);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_queueOut, queueOut, THREAD_NUM, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_neighboursPrefixSum, THREAD_NUM);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_neighboursPrefixSum, neighboursPrefixSum, THREAD_NUM, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_visited, verticesCount);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_visited, visited, verticesCount, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_totalNeighbours, 1);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_totalNeighbours, &totalNeighbours, 1, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_neighbourCounts, THREAD_NUM);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_neighbourCounts, neighbourCounts, THREAD_NUM, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

#pragma endregion

	while (totalNeighbours > 0)
	{
		gatherScan << <1, THREAD_NUM >> > (dev_queueIn, dev_queueOut, dev_c, dev_r,
			dev_neighboursPrefixSum, dev_visited, dev_totalNeighbours, dev_neighbourCounts, newColor, startingColor);

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching gatherScan!\n", cudaStatus);
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "gatherScan launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = deviceMemcpy(queueOut, dev_queueOut, THREAD_NUM, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "gueueOut dev to host memcpy failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = deviceMemcpy(dev_queueIn, dev_queueOut, THREAD_NUM, hipMemcpyDeviceToDevice);
		cudaStatus = deviceMemcpy(neighboursPrefixSum, dev_neighbourCounts, THREAD_NUM, hipMemcpyDeviceToHost);

		thrust::exclusive_scan(neighboursPrefixSum, neighboursPrefixSum + totalNeighbours + 1, neighboursPrefixSum);
		totalNeighbours = neighboursPrefixSum[totalNeighbours];

		cudaStatus = deviceMemcpy(dev_neighboursPrefixSum, neighboursPrefixSum, THREAD_NUM, hipMemcpyHostToDevice);
		cudaStatus = deviceMemcpy(dev_totalNeighbours, &totalNeighbours, 1, hipMemcpyHostToDevice);

	}
	cudaStatus = deviceMemcpy(visited, dev_visited, verticesCount, hipMemcpyDeviceToHost);

Error:
	/*hipFree(dev_c); dfdfv
	hipFree(dev_r);
	hipFree(dev_queueIn);
	hipFree(dev_queueOut);
	hipFree(dev_neighboursPrefixSum);
	hipFree(dev_visited);
	hipFree(dev_totalNeighbours);
	hipFree(dev_neighbourCounts);*/ 

	return visited;
}