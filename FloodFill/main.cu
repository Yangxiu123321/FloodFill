#include "floodFillWithCpu.h"
#include "hostFunctions.h"
#include "deviceFunctions.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/scan.h>

#include <stdlib.h>
#include <stdio.h>
using namespace std;
void FloodFillWithGPU();

int main()
{
	int height = 1;
	int width = 5;

	//int** array = new int*[height];
	//for (int i = 0; i < height; i++)
	//{
	//	array[i] = new int[width];
	//	for (int j = 0; j < width; j++)
	//		array[i][j] = 0;
	//}
	//array[0][3] = 4;
	////floodFillWithCpu(array, height, width, 3, 2, 1);

	//bfs(array, height, width, 0, 0, 1);

	//for (int i = 0; i < height; i++)
	//{
	//	for (int j = 0; j < width; j++)
	//		cout << array[i][j] << " ";
	//	cout << endl;
	//}
	FloodFillWithGPU();
}

void FloodFillWithGPU()
{
	//int* c = (int*)malloc(sizeof(int) * 15);
	int verticesCount = 16;
	int c[15] = { 2, 3, 4, 5, 6, 7, 8 , 9, 10, 11, 12, 13, 14, 15, 16 };
	int r[18] = { 0, 0, 3, 6, 13, 15, 15, 15, 15, 15, 15, 15, 15, 15, 15, 15, 15, 15 };
	int queueIn[10] = { 2, 3, 4, -1, -1, -1, -1, -1, -1, -1 };
	int queueOut[12] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int neighboursPrefixSum[12] = { 3, 7, 2, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int visited[17];
	int totalNeighbours;
	int neighbourCounts[17];

	for (int i = 0; i < 17; i++)
		visited[i] = 0;


	thrust::exclusive_scan(neighboursPrefixSum, neighboursPrefixSum + 12, neighboursPrefixSum);
	totalNeighbours = neighboursPrefixSum[11];
	printf("%d\n", totalNeighbours);


	//device arrays
	int* dev_verticesCount;
	int* dev_c = 0;
	int* dev_r;
	int* dev_queueIn;
	int* dev_queueOut;
	int* dev_neighboursPrefixSum;
	int* dev_visited = 0;
	int* dev_totalNeighbours;
	int* dev_neighbourCounts;

	hipError_t cudaStatus;

#pragma region Mallocs
	cudaStatus = deviceMalloc(&dev_c, 15);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_c, c, 15, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_r, 18);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_r, r, 18, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_queueIn, 10);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_queueIn, queueIn, 10, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_queueOut, 12);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_queueOut, queueOut, 12, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_neighboursPrefixSum, 12);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_neighboursPrefixSum, neighboursPrefixSum, 12, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_visited, 17);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_visited, visited, 17, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_totalNeighbours, 1);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_totalNeighbours, &totalNeighbours, 1, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_neighbourCounts, 17);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_neighbourCounts, neighbourCounts, 17, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

#pragma endregion



	gatherScan << <1, THREAD_NUM >> > (dev_queueIn, dev_queueOut, dev_c, dev_r, 
		dev_neighboursPrefixSum, dev_visited, dev_totalNeighbours, dev_neighbourCounts);
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching scatterKernel!\n", cudaStatus);
		goto Error;
	}
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "calculateBackwardMask launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = deviceMemcpy(queueOut, dev_queueOut, 12, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "calculateBackwardMask launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	for (int i = 0; i < 12; i++)
		printf("%d ", queueOut[i]);
Error:
	hipFree(dev_c);
}