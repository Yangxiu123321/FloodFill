#include "floodFillWithCpu.h"
#include "hostFunctions.h"
#include "deviceFunctions.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/scan.h>

#include <stdlib.h>
#include <stdio.h>
using namespace std;
void FloodFillWithGPU();

int main()
{
	int height = 1;
	int width = 5;

	//int** array = new int*[height];
	//for (int i = 0; i < height; i++)
	//{
	//	array[i] = new int[width];
	//	for (int j = 0; j < width; j++)
	//		array[i][j] = 0;
	//}
	//array[0][3] = 4;
	////floodFillWithCpu(array, height, width, 3, 2, 1);

	//bfs(array, height, width, 0, 0, 1);

	//for (int i = 0; i < height; i++)
	//{
	//	for (int j = 0; j < width; j++)
	//		cout << array[i][j] << " ";
	//	cout << endl;
	//}
	FloodFillWithGPU();
}

void FloodFillWithGPU()
{
	//int* c = (int*)malloc(sizeof(int) * 15);
	int verticesCount = 16;
	int c[16] = { 2, 3, 4, 5, 6, 7, 8 , 9, 10, 11, 12, 13, 14, 15, 16, 11 };
	int r[18] = { 0, 0, 3, 6, 13, 15, 16, 16, 16, 16, 16, 16, 16, 16, 16, 16, 16, 16 };
	int queueIn[13] = { 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 };
	int queueOut[13] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int neighboursPrefixSum[17] = { 3, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int visited[17];
	int totalNeighbours;
	int neighbourCounts[17];

	for (int i = 0; i < 17; i++)
		visited[i] = 0;


	thrust::exclusive_scan(neighboursPrefixSum, neighboursPrefixSum + 17, neighboursPrefixSum);
	totalNeighbours = neighboursPrefixSum[16];
	printf("total neighbours count: %d\n", totalNeighbours);


	//device arrays
	int* dev_verticesCount;
	int* dev_c = 0;
	int* dev_r;
	int* dev_queueIn;
	int* dev_queueOut;
	int* dev_neighboursPrefixSum;
	int* dev_visited = 0;
	int* dev_totalNeighbours;
	int* dev_neighbourCounts;

	hipError_t cudaStatus;

#pragma region Mallocs
	cudaStatus = deviceMalloc(&dev_c, 16);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_c, c, 16, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_r, 18);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_r, r, 18, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_queueIn, 13);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_queueIn, queueIn, 13, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_queueOut, 13);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_queueOut, queueOut, 13, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_neighboursPrefixSum, 17);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_neighboursPrefixSum, neighboursPrefixSum, 17, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_visited, 17);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_visited, visited, 17, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_totalNeighbours, 1);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_totalNeighbours, &totalNeighbours, 1, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_neighbourCounts, 17);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_neighbourCounts, neighbourCounts, 17, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

#pragma endregion

	int i = 3;
	while (totalNeighbours > 0)
	{
		gatherScan << <1, THREAD_NUM >> > (dev_queueIn, dev_queueOut, dev_c, dev_r,
			dev_neighboursPrefixSum, dev_visited, dev_totalNeighbours, dev_neighbourCounts);

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching gatherScan!\n", cudaStatus);
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "gatherScan launch failed: %s\n", hipGetErrorString(cudaStatus));
			//goto Error;
		}

		cudaStatus = deviceMemcpy(queueOut, dev_queueOut, 13, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "gueueOut dev to host memcpy failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		for (int i = 0; i < 12; i++)
			printf("%d ", queueOut[i]);

		printf("\n");

		cudaStatus = deviceMemcpy(dev_queueIn, dev_queueOut, 13, hipMemcpyDeviceToDevice);
		cudaStatus = deviceMemcpy(neighboursPrefixSum, dev_neighbourCounts, 17, hipMemcpyDeviceToHost);

		printf("neighbour counts befere scan");
		for (int i = 0; i < 17; i++)
			printf("%d ", neighboursPrefixSum[i]);

		printf("\n");
		thrust::exclusive_scan(neighboursPrefixSum, neighboursPrefixSum + totalNeighbours + 1, neighboursPrefixSum);
		totalNeighbours = neighboursPrefixSum[totalNeighbours];

		printf("neighbour counts befere scan");
		for (int i = 0; i < 17; i++)
			printf("%d ", neighboursPrefixSum[i]);

		printf("\n"); printf("total neighbours count: %d\n", totalNeighbours);

		cudaStatus = deviceMemcpy(dev_neighboursPrefixSum, neighboursPrefixSum, 17, hipMemcpyHostToDevice);
		cudaStatus = deviceMemcpy(dev_totalNeighbours, &totalNeighbours, 1, hipMemcpyHostToDevice);

	Error:
		hipFree(dev_c);
	}
}