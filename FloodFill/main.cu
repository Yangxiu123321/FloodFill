#include "floodFillWithCpu.h"
#include "floodFillWithGpu.h"
#include "hostFunctions.h"
#include "deviceFunctions.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/scan.h>

#include <stdlib.h>
#include <stdio.h>
using namespace std;
int* FloodFillWithGPU(int* arr, int rows, int cols, int xStarting, int yStarting, int newColor);

int main()
{
	int xStarting = 0, yStarting = 1;
	int newColor = 2;

	int arr1[9] = {		0, 0, 1,
						0, 0, 0,
						0, 0, 0 };

	int arr2[9] ={	0, 0, 1,
					0, 1, 1,
					0, 1, 0 };

	int arr3[36] = {	0, 0, 1, 0, 0, 0,
						0, 0, 0, 1, 0, 0,
						0, 0, 5, 5, 3, 0, 
						0, 0, 5, 5, 3, 0,
						0, 1, 5, 5, 0, 0,
						0, 1, 0, 0, 0, 0 };

	int arr4[36] = {	0, 0, 0, 0, 0, 0,
						0, 1, 1, 1, 1, 0,
						0, 1, 0, 0, 3, 0,
						0, 0, 3, 0, 3, 2,
						0, 0, 1, 1, 0, 0,
						0, 0, 0, 0, 0, 0,	};

	int arr5[42] = {	0, 1, 1, 1, 1, 0,
						0, 0, 1, 1, 1, 1,
						1, 0, 1, 1, 1, 1,
						1, 0, 1, 1, 1, 1,
						1, 0, 0, 0, 0, 1,
						1, 1, 1, 1, 0, 0, };

	int testRows[5] = { 3, 3, 6, 6, 7 };
	int testCols[5] = { 3, 3, 6, 6, 6 };
	int* testArrays[5] = { arr1, arr2, arr3, arr4, arr5 };

	for (int i = 0; i < 4; i++)
	{
		int rows = testRows[i], cols = testCols[i];
		int* arr = testArrays[i];

		printf("Starting point: (0, 1)\n");
		printf("New color: 2 \n");
		printf("\n");
		printf("Array before filling: \n");

		for (int i = 0, v = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
				printf("%d ", arr[v++]);
			printf("\n");
		}
		printf("\n");

		int* colouredyGpu = FloodFillWithGPU(arr, rows, cols, xStarting, yStarting, newColor);

		printf("Array after filling by GPU: \n");

		for (int i = 0, v = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
				printf("%d ", colouredyGpu[v++]);
			printf("\n");
		}
		printf("\n");

		int* colouredByCpu = floodFillWithCpu(arr, rows, cols, xStarting, yStarting, newColor);

		printf("Array after filling by CPU: \n");

		int equal = 1;
		for (int i = 0, v = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
			{
				if (colouredByCpu[v] != colouredyGpu[v])
				{
					equal = 0;
				}
				printf("%d ", colouredByCpu[v++]);
			}
			printf("\n");
		}
			if(equal)
				printf("Arrays are equal! \n");
			else
				printf("Arrays are not equal! \n");


		printf("\n");
		printf("***********************************\n");
		printf("\n");
	}
}
