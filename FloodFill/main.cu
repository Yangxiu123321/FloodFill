#include "floodFillWithCpu.h"
#include "hostFunctions.h"
#include "deviceFunctions.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/scan.h>

#include <stdlib.h>
#include <stdio.h>
using namespace std;
int* FloodFillWithGPU(int* arr, int rows, int cols, int xStarting, int yStarting, int newColor);

int main()
{
	int xStarting = 0, yStarting = 1;
	int newColor = 2;

	int arr1[9] = {		0, 0, 1,
						0, 0, 0,
						0, 0, 0 };

	int arr2[9] ={	0, 0, 1,
					0, 1, 1,
					0, 1, 0 };

	int arr3[36] = {	0, 0, 1, 0, 0, 0,
						0, 0, 0, 1, 0, 0,
						0, 0, 5, 5, 3, 0, 
						0, 0, 5, 5, 3, 0,
						0, 1, 5, 5, 0, 0,
						0, 1, 0, 0, 0, 0 };

	int arr4[36] = {	0, 0, 0, 0, 0, 0,
						0, 1, 1, 1, 1, 0,
						0, 1, 0, 0, 3, 0,
						0, 0, 3, 0, 3, 2,
						0, 0, 1, 1, 0, 0,
						0, 0, 0, 0, 0, 0,	};

	int testRows[4] = { 3, 3, 6, 6 };
	int testCols[4] = { 3, 3, 6, 6 };
	int* testArrays[4] = { arr1, arr2, arr3, arr4 };

	for (int i = 0; i < 4; i++)
	{
		int rows = testRows[i], cols = testCols[i];
		int* arr = testArrays[i];

		printf("Starting point: (0, 1)\n");
		printf("New color: 1 \n");
		printf("\n");
		printf("Array before filling: \n");

		for (int i = 0, v = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
				printf("%d ", arr[v++]);
			printf("\n");
		}
		
		int* coloured = FloodFillWithGPU(arr, rows, cols, xStarting, yStarting, newColor);

		printf("Array after filling: \n");

		for (int i = 0, v = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
				printf("%d ", coloured[v++]);
			printf("\n");
		}
		printf("\n");
		printf("***********************************\n");
		printf("\n");

		//int** array = new int*[height];
		//for (int i = 0; i < height; i++)
		//{
		//	array[i] = new int[width];
		//	for (int j = 0; j < width; j++)
		//		array[i][j] = 0;
		//}
		//array[0][3] = 4;
		////floodFillWithCpu(array, height, width, 3, 2, 1);

		//bfs(array, height, width, 0, 0, 1);

		//for (int i = 0; i < height; i++)
		//{
		//	for (int j = 0; j < width; j++)
		//		cout << array[i][j] << " ";
		//	cout << endl;
		//}

	}
}

int* FloodFillWithGPU(int* arr, int rows, int cols, int xStarting, int yStarting, int newColor)
{
	int* r;
	int* c;
	int startingVertex = yStarting * cols + xStarting;
	int startingColor = arr[startingVertex];
	int edgesCount = getEdgesCount(rows, cols);
	int verticesCount = rows * cols;

	prepareArrays(arr, &r, &c, rows, cols);

	//int c[16] = { 2, 3, 4, 5, 6, 7, 8 , 9, 10, 11, 12, 13, 14, 15, 16, 11 };
	//int r[18] = { 0, 0, 3, 6, 13, 15, 16, 16, 16, 16, 16, 16, 16, 16, 16, 16, 16, 16 };
	int* queueIn = (int*)malloc(THREAD_NUM * sizeof(int));// = { 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 };
	int* queueOut = (int*)malloc(THREAD_NUM * sizeof(int));// = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int* neighboursPrefixSum = (int*)malloc(THREAD_NUM * sizeof(int));// = { 3, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int* visited = (int*)malloc(rows * cols * sizeof(int));
	int totalNeighbours;
	int* neighbourCounts = (int*)malloc(THREAD_NUM * sizeof(int));

	for (int i = 0; i < THREAD_NUM; i++)
	{
		queueIn[i] = -1;
		queueOut[i] = 0;
		neighboursPrefixSum[i] = 0;
	}
	queueIn[0] = startingVertex;
	neighboursPrefixSum[0] = r[startingVertex + 1] - r[startingVertex];

	for (int i = 0; i < rows * cols; i++)
		visited[i] = arr[i];


	thrust::exclusive_scan(neighboursPrefixSum, neighboursPrefixSum + THREAD_NUM, neighboursPrefixSum);
	totalNeighbours = neighboursPrefixSum[THREAD_NUM - 1];
	//printf("total neighbours count: %d\n", totalNeighbours);

	hipError_t cudaStatus;

#pragma region Device arrays

	int* dev_verticesCount;
	int* dev_c = 0;
	int* dev_r;
	int* dev_queueIn;
	int* dev_queueOut;
	int* dev_neighboursPrefixSum;
	int* dev_visited = 0;
	int* dev_totalNeighbours;
	int* dev_neighbourCounts;

#pragma endregion

#pragma region Mallocs
	cudaStatus = deviceMalloc(&dev_c, edgesCount);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_c, c, edgesCount, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_r, (verticesCount + 1));
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_r, r, (verticesCount + 1), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_queueIn, THREAD_NUM);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_queueIn, queueIn, THREAD_NUM, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_queueOut, THREAD_NUM);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_queueOut, queueOut, THREAD_NUM, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_neighboursPrefixSum, THREAD_NUM);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_neighboursPrefixSum, neighboursPrefixSum, THREAD_NUM, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_visited, verticesCount);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_visited, visited, verticesCount, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_totalNeighbours, 1);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_totalNeighbours, &totalNeighbours, 1, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

	cudaStatus = deviceMalloc(&dev_neighbourCounts, THREAD_NUM);
	if (cudaStatus != hipSuccess) goto Error;
	cudaStatus = deviceMemcpy(dev_neighbourCounts, neighbourCounts, THREAD_NUM, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto Error;

#pragma endregion


	while (totalNeighbours > 0)
	{
		gatherScan << <1, THREAD_NUM >> > (dev_queueIn, dev_queueOut, dev_c, dev_r,
			dev_neighboursPrefixSum, dev_visited, dev_totalNeighbours, dev_neighbourCounts, newColor, startingColor);

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching gatherScan!\n", cudaStatus);
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "gatherScan launch failed: %s\n", hipGetErrorString(cudaStatus));
			//goto Error;
		}

		cudaStatus = deviceMemcpy(queueOut, dev_queueOut, THREAD_NUM, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "gueueOut dev to host memcpy failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		/*for (int i = 0; i < THREAD_NUM; i++)
			printf("%d ", queueOut[i]);

		printf("\n");*/

		cudaStatus = deviceMemcpy(dev_queueIn, dev_queueOut, THREAD_NUM, hipMemcpyDeviceToDevice);
		cudaStatus = deviceMemcpy(neighboursPrefixSum, dev_neighbourCounts, THREAD_NUM, hipMemcpyDeviceToHost);

		/*printf("neighbour counts befere scan \n");
		for (int i = 0; i < THREAD_NUM; i++)
			printf("%d ", neighboursPrefixSum[i]);

		printf("\n");*/
		thrust::exclusive_scan(neighboursPrefixSum, neighboursPrefixSum + totalNeighbours + 1, neighboursPrefixSum);
		totalNeighbours = neighboursPrefixSum[totalNeighbours];

		/*printf("neighbour counts after scan \n");
		for (int i = 0; i < THREAD_NUM; i++)
			printf("%d ", neighboursPrefixSum[i]);

		printf("\n"); printf("total neighbours count: %d\n", totalNeighbours);
		printf("\n");
		printf("\n");*/

		cudaStatus = deviceMemcpy(dev_neighboursPrefixSum, neighboursPrefixSum, THREAD_NUM, hipMemcpyHostToDevice);
		cudaStatus = deviceMemcpy(dev_totalNeighbours, &totalNeighbours, 1, hipMemcpyHostToDevice);
	
	}
Error:
		hipFree(dev_c);
	cudaStatus = deviceMemcpy(visited, dev_visited, verticesCount, hipMemcpyDeviceToHost);

	return visited;
}